// Copyright Supranational LLC
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0

#include <hip/hip_runtime.h>

#include <ec/jacobian_t.hpp>
#include <ec/xyzz_t.hpp>

#include <ff/pasta.hpp>

typedef jacobian_t<pallas_t> point_t;
typedef xyzz_t<pallas_t> bucket_t;
typedef bucket_t::affine_t affine_t;
typedef vesta_t scalar_t;

#include <msm/pippenger.cuh>

#ifndef __CUDA_ARCH__
extern "C"
RustError cuda_pippenger_pallas(point_t *out, const affine_t points[], size_t npoints,
                                              const scalar_t scalars[])
{   return mult_pippenger<bucket_t>(out, points, npoints, scalars);   }
#endif
