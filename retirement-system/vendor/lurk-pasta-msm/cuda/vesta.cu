// Copyright Supranational LLC
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0

#include <hip/hip_runtime.h>

#include <ec/jacobian_t.hpp>
#include <ec/xyzz_t.hpp>

#include <ff/pasta.hpp>

typedef jacobian_t<vesta_t> point_t;
typedef xyzz_t<vesta_t> bucket_t;
typedef bucket_t::affine_t affine_t;
typedef pallas_t scalar_t;

#include <msm/pippenger.cuh>

#ifndef __CUDA_ARCH__
extern "C"
RustError cuda_pippenger_vesta(point_t *out, const affine_t points[], size_t npoints,
                                              const scalar_t scalars[])
{   return mult_pippenger<bucket_t>(out, points, npoints, scalars);   }
#endif
